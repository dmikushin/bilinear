#include "hip/hip_runtime.h"
#include <chrono>
#include <iomanip>
#include <stdio.h>
#include <math.h>
#include <iostream>
#include <vector>
#include "EasyBMP.h"
#include "gpu.h"

using namespace std;
using namespace std::chrono;

#define ARR(T, i, j) (T[(i) + (j) * width])

#if (defined(_MSC_VER) && defined(_WIN64)) || defined(__LP64__)
#define __RET_PTR   "l"
#else
#define __RET_PTR   "r"
#endif

union RGBApixel_
{
	RGBApixel p;
	int i;
};

static __device__ __inline__ void interpolate(
	const RGBApixel* pixels, RGBApixel* output, int width, float x, float y)
{
	int px = (int)x; // floor of x
	int py = (int)y; // floor of y
	const int stride = width;
	const RGBApixel* p0 = &pixels[0] + px + py * stride; // pointer to first pixel

	// Load the four neighboring pixels
	RGBApixel_ p1_; p1_.i = *(int*)&p0[0 + 0 * stride];
	RGBApixel_ p2_; p2_.i = *(int*)&p0[1 + 0 * stride];
	RGBApixel_ p3_; p3_.i = *(int*)&p0[0 + 1 * stride];
	RGBApixel_ p4_; p4_.i = *(int*)&p0[1 + 1 * stride];

	const RGBApixel& p1 = p1_.p;
	const RGBApixel& p2 = p2_.p;
	const RGBApixel& p3 = p3_.p;
	const RGBApixel& p4 = p4_.p;
	
	// Calculate the weights for each pixel
	float fx = x - px;
	float fy = y - py;
	float fx1 = 1.0f - fx;
	float fy1 = 1.0f - fy;

	int w1 = fx1 * fy1 * 256.0f + 0.5f;
	int w2 = fx  * fy1 * 256.0f + 0.5f;
	int w3 = fx1 * fy  * 256.0f + 0.5f;
	int w4 = fx  * fy  * 256.0f + 0.5f;

	// Calculate the weighted sum of pixels (for each color channel)
	int outr = p1.Red * w1 + p2.Red * w2 + p3.Red * w3 + p4.Red * w4;
	int outg = p1.Green * w1 + p2.Green * w2 + p3.Green * w3 + p4.Green * w4;
	int outb = p1.Blue * w1 + p2.Blue * w2 + p3.Blue * w3 + p4.Blue * w4;
	int outa = p1.Alpha * w1 + p2.Alpha * w2 + p3.Alpha * w3 + p4.Alpha * w4;

	RGBApixel ret;
	ret.Red = (outr + 128) >> 8;
	ret.Green = (outg + 128) >> 8;
	ret.Blue = (outb + 128) >> 8;
	ret.Alpha = (outa + 128) >> 8;

	RGBApixel_* output_ = (RGBApixel_*)output;
	output_->p = ret;
}

__global__ void bilinear (const int width, const int height,
	RGBApixel* input, RGBApixel* output)
{
	int j = blockDim.y * blockIdx.y + threadIdx.y;
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	
	if (j >= 2 * height) return;
	if (i >= 2 * width) return;

	float x = width * (i - 0.5f) / (float)(2 * width);
	float y = height * (j - 0.5f) / (float)(2 * height);

	interpolate(input, &output[i + j * 2 * width], width, x, y);
}

int main(int argc, char* argv[])
{
	if (argc != 2)
	{
		cout << "Usage: " << argv[0] << " <filename>" << endl;
		return 0;
	}

	char* filename = argv[1];

	BMP AnImage;
	AnImage.ReadFromFile(filename);
	int width = AnImage.TellWidth();
	int height = AnImage.TellHeight();

	vector<RGBApixel> input(width * (height + 1) + 1);
	vector<RGBApixel> output(4 * width * height);
	for (int i = 0; i < width; i++)
		for (int j = 0; j < height; j++)
			input[i + j * width] = AnImage.GetPixel(i, j);
	memset(&input[height * width], 0, (width + 1) * sizeof(RGBApixel));

	RGBApixel *dinput, *doutput;
	GPU_CALL(gpuMalloc(&dinput, sizeof(RGBApixel) * input.size()));
	GPU_CALL(gpuMalloc(&doutput, sizeof(RGBApixel) * output.size()));
	GPU_CALL(gpuMemcpy(dinput, &input[0], sizeof(RGBApixel) * input.size(), gpuMemcpyHostToDevice));

	auto start = high_resolution_clock::now();

	dim3 szblock(128, 1, 1);
	dim3 nblocks(2 * width / szblock.x, 2 * height, 1);
	if (2 * width % szblock.x) nblocks.x++;
	bilinear<<<nblocks, szblock>>>(width, height, dinput, doutput);
	GPU_CALL(gpuGetLastError());
	GPU_CALL(gpuDeviceSynchronize());

	auto finish = high_resolution_clock::now();

	cout << "GPU kernel time = " <<
		duration_cast<milliseconds>(finish - start).count() <<
		" ms" << endl;

	GPU_CALL(gpuMemcpy(&output[0], doutput, sizeof(RGBApixel) * output.size(), gpuMemcpyDeviceToHost));

	AnImage.SetSize(2 * width, 2 * height);
	for (int i = 0; i < 2 * width; i++)
		for (int j = 0; j < 2 * height; j++)
			AnImage.SetPixel(i, j, output[i + j * 2 * width]);

	AnImage.WriteToFile("output_gpu.bmp");

	GPU_CALL(gpuFree(dinput));
	GPU_CALL(gpuFree(doutput));
 
	return 0;
}

